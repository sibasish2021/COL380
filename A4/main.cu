#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include<fstream>
#define endl "\n"
#define uint uint32_t
#define int int32_t
#define maxvalue UINT32_MAX
using namespace std;

__device__ __managed__ int n,m,k1,k2;
// __device__ __managed__ uint64_t maxvalue=((uint64_t)(1e32)-1);

void inputParameters(int* klocal,string fileName)
{
    ifstream f1(fileName);
    f1.read((char *)&n,sizeof(int));
    f1.read((char *)&m,sizeof(int));
    f1.read((char *)klocal,sizeof(int));
    f1.close();
}

void InputData(string fileName,int* matrix,uint* matrixdata,int* klocal)
{
    ifstream f1(fileName);
    f1.read((char *)&n,sizeof(int));
    f1.read((char *)&m,sizeof(int));
    f1.read((char *)klocal,sizeof(int));

    // cout<<"values of n,m,k are"<<n<<" "<<m<<" "<<*klocal<<endl;
    int k=*klocal;
    

    for(int i=0;i<n/m;i++)
    {
        for(int j=0;j<n/m;j++)
        {
            matrix[i*n/m+j]=-1;
        }
    }

    for(int i=0;i<k;i++)
    {
        int posx,posy;
        f1.read((char *)&posx,sizeof(int));
        f1.read((char *)&posy,sizeof(int));
        matrix[posx*n/m+posy]=i;
        // cout<<"Check3"<<endl;
        for(int x=0;x<m;x++)
        {
            for(int y=0;y<m;y++)
            {
                uint16_t temp;
                f1.read((char *)&temp,sizeof(uint16_t));
                matrixdata[i*m*m+x*m+y]=(uint) temp;
                // if(posx==11 && posy==19)
                //     cout<<"Value at position "<< x <<" "<<y<<" is "<<matrixdata[i*m*m+x*m+y]<<endl;
            }
        }
    }
    // cout<<"Check4"<<endl;
    f1.close();   
}

void printmatrix(int* matrix,uint* matrixdata,int* k)
{
    cout<<"n,m,k are"<<n<<" "<<m<<" "<<*k<<endl;

    for(int i=0;i<n/m;i++)
    {
        for(int j=0;j<n/m;j++)
        {
            cout<<matrix[i*n/m+j]<<" ";
        }
        cout<<endl;
    }
}


void outputData(string fileName,int* matrix,uint* matrixdata,int* klocal)
{
    // cout<<"values of n,m,k in output are "<<n<<" "<<m<<" "<<*klocal<<endl;
    ofstream f1(fileName,ios::binary);
    f1.write((char *)&n,sizeof(int32_t));
    f1.write((char *)&m,sizeof(int32_t));
    f1.write((char *)klocal,sizeof(int32_t));
    // int k=*klocal;

    for(int i=0;i<(n/m)*(n/m);i++)
    {
        if(matrix[i]<0)
            continue;
        int posx=i/(n/m);
        int posy=i%(n/m);    
        f1.write((char *)&posx,sizeof(int32_t));
        f1.write((char *)&posy,sizeof(int32_t));
        for(int j=0;j<m;j++)
        {
            for(int k=0;k<m;k++)
            {
                uint temp=(uint)(matrixdata[matrix[i]*m*m+j*m+k]);
                // if(posx==0 && posy==1)
                //     cout<<"Values are "<<j<<" "<<k<<" "<<temp<<endl; 
                f1.write((char *)&temp,sizeof(uint32_t));
            }
        }
    }
    f1.close();
}

__device__ void blockMult(uint* a,uint* b,uint* c)
{
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<m;j++)
        {
            uint64_t ans=c[i*m+j];
            for(int k=0;k<m;k++)
            {
                ans+=(uint64_t)(a[i*m+k]*b[k*m+j]);
                if (ans>(uint64_t)maxvalue) 
                {
                    ans=maxvalue;
                }
            }
            c[i*m+j]=ans;
        }
    }
}

__global__ void matrixMult(int* a,int* b,int* c,uint* aData,uint* bData,uint* cData)
{
    int blockNum=blockIdx.x*blockDim.x+threadIdx.x;
    if(blockNum >= (n/m)*(n/m)) 
    {
        return;
    }
    int posx= blockNum/(n/m);
    int posy= blockNum%(n/m);
    for (int i=0;i<n/m;i++)
    {
        if ((a[posx*(n/m)+i]<0)||(b[i*(n/m)+posy]<0)) 
        {
            continue;
        }
        blockMult(aData+(uint)a[posx*(n/m)+i]*m*m,bData+(uint)b[i*(n/m)+posy]*m*m, cData+(uint)c[blockNum]*m*m);
    }
    bool check = true;
    for (int i=0;i<m*m;i++) 
    {
        if(*(cData+(uint)c[blockNum]*m*m+i) != 0)
        {
            check=false;
            break;
        }
    }
    if (check) 
    {
        c[blockNum]=-1;
    }
}

signed main(int argc, char *argv[])
{
    ios_base::sync_with_stdio(0);
    cin.tie(0);
    cout.tie(0);
    
    string inputFile1(argv[1]);
    string inputFile2(argv[2]);
    string outputFile(argv[3]);

    int* a;int* b;int* c;
    int* ga;int* gb;int* gc;
    uint* aData;uint* bData;uint* cData;
    uint* gaData;uint* gbData;uint* gcData;
    
    inputParameters(&k1,inputFile1);
    a=(int*)calloc(n/m*n/m,sizeof(int));
    aData=(uint*)calloc(m*m*k1,sizeof(uint));
    
    inputParameters(&k2,inputFile2);
    b=(int*)calloc(n/m*n/m,sizeof(int));
    bData=(uint*)calloc(m*m*k2,sizeof(uint));
    
    InputData(inputFile1,a,aData,&k1);
    InputData(inputFile2,b,bData,&k2);
    
    c=(int*)calloc((n/m)*(n/m),sizeof(int));
    cData=(uint*)calloc(n*n,sizeof(uint));
    for(int i=0;i<(n/m)*(n/m);i++)
    {
        c[i]=i;
    }

    hipMalloc((void**)&ga, (n/m)*(n/m)* sizeof(int));
    hipMalloc((void**)&gb, (n/m)*(n/m)* sizeof(int));
    hipMalloc((void**)&gc, (n/m)*(n/m)* sizeof(int));
    
    hipMalloc((void**)&gaData, k1*m*m* sizeof(uint));
    hipMalloc((void**)&gbData, k2*m*m* sizeof(uint));
    hipMalloc((void**)&gcData, n*n* sizeof(uint));

    int dev = -1;
    hipGetDevice(&dev);
    hipMemPrefetchAsync(&n, sizeof(int), dev);
    hipMemPrefetchAsync(&m, sizeof(int), dev);

    hipMemcpyAsync(ga, a, (n/m)*(n/m)*sizeof(int),hipMemcpyHostToDevice,0);
    hipMemcpyAsync(gb, b, (n/m)*(n/m)*sizeof(int),hipMemcpyHostToDevice,0);
    hipMemcpyAsync(gc, c, (n/m)*(n/m)*sizeof(int),hipMemcpyHostToDevice,0);
    
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipStream_t stream2;
    hipStreamCreate(&stream2);
    hipStream_t stream3;
    hipStreamCreate(&stream3);

    hipMemcpyAsync(gaData,aData,k1*m*m*sizeof(uint),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(gbData,bData,k2*m*m*sizeof(uint),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(gcData,cData,n*n*sizeof(uint),hipMemcpyHostToDevice,stream3);
    
    // hipStreamSynchronize(0);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);

    int threads_per_block=1024;
    int blocks_per_grid = ((n/m)*(n/m)+threads_per_block-1)/threads_per_block;
    
    dim3 dimGrid(blocks_per_grid,1,1);
    dim3 dimBlock(threads_per_block,1,1);
    
    hipEvent_t	event1,	event2;
    hipEventCreate(&event1);	
    hipEventCreate(&event2);
    hipEventRecord(event1,0);
    matrixMult<<<dimGrid,dimBlock,0>>>(ga,gb,gc,gaData,gbData,gcData);
    hipEventRecord(event2,0);
    hipEventSynchronize(event2);
    float ms;
    hipEventElapsedTime(&ms,event1,event2);
    // cout<<"Time taken by the kernel is "<<ms<<endl;

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    { 
        printf("Error: %s\n", hipGetErrorString(err));
    }
    
    // hipDeviceSynchronize();

    hipMemcpy(c,gc, (n/m)*(n/m)*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(cData,gcData,n*n*sizeof(uint),hipMemcpyDeviceToHost);
    // printBlocksCPU(c,cData,n,m);

    int k3=0;
    for(int i=0;i<n/m;i++)
    {
        for(int j=0;j<n/m;j++)
        {
            if(c[i*(n/m)+j]>=0)
            {
                k3++;
            }
        }
    }

    outputData(outputFile,c,cData,&k3);
    
    hipFree(ga);
    hipFree(gb);
    hipFree(gc);
    hipFree(gaData);
    hipFree(gbData);
    hipFree(gcData);
    
    free(a);
    free(b);
    free(c);
    free(aData);
    free(bData);
    free(cData);

    return 0;
}